/**
 * File: FDTD2Dsolver.c
 *
 * Direct convertion from MATLAB script file linearadvectionFOU2D.m,
 * which is also uploaded to the project.
 *
 * Following is the original file document from the MATLAB script:
 *
 * Description: Solves the 2D linear advection equation
 * dU/dt + vx dU/dx + vy dU/dy = 0,
 * using first order forward difference in time
 * and first order backward differences in space.
 * The solution is calculated over [p, q] x [r, s] using NX, NY points
 * in the x and y directions respectively and plotted
 * after ntimesteps time steps, i.e. the final
 * solution is at time, ntimesteps*dt seconds.
 *
 * Boundary conditions: Dirichlet boundary conditions are used everywhere.
 *
 * Subfunction: gaussian2D
 *
 * Note:
 * This 2D problem has an analytical solution.
 * If initial condition is U(x,0) = f(x, y) then it can be shown
 * that the exact solution is U(x, y, t) = f(x - vx t, y - vy t).
 * i.e. the initial state is translated (advected) in the x and y directions
 * with speeds vx and vy respectively.
 *
 * Stability analysis for the timestep is very complicated so a heuristic
 * formula has been used based on the 1D case and a safety factor, F.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>  /* file writing */
#include <string.h> /* memcpy */
#include <math.h>   /* exp */
#include <stdlib.h> /* malloc */
#include <time.h>   /* timing */

#define MIN(A, B) ((A) < (B) ? (A) : (B))

/* p, q, r, s specify size of domain */
#define p  0.0f
#define q  100.0f
#define r  0.0f
#define s  100.0f
/* water speed in x direction */
#define vx  0.5f
/* water speed in y direction */
#define vy  0.5f
/* cenx, ceny centre for Gaussian */
#define cenx ((p) + (q)) / 2
#define ceny ((r) + (s)) / 2
/* rad is Gaussian 'radius' */
#define rad 20.0f


/**
 * 2 dimensional Gaussian function over [p, q]x[r, s], height 1, centred on
 * (cenx, ceny) which becomes zero rad away from the centre.
 */
__device__ float gaussian2D(float x, float y)
{
    /* square of distance from centre */
    float d2 = (x - cenx) * (x - cenx) + (y - ceny) * (y - ceny);
    /* value of Gaussian function at (x, y) */
    return (d2 < rad * rad) ? exp(-0.01 * d2) : 0;
}

__global__ void iteration(float *u, float *u0, float Cx, float Cy, int NX, int NY)
{
    int ix0 = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int iy0 = blockIdx.y * blockDim.y + threadIdx.y + 1;

    int stride_x = blockDim.x * gridDim.x;
    int stride_y = blockDim.y * gridDim.y;

    for(int ix = ix0; ix < NX; ix += stride_x)
        for(int iy = iy0; iy < NY; iy += stride_y)
        {
            u[iy * NX + ix] = u0[iy * NX + ix]
                - Cx * (u0[iy * NX + ix] - u0[iy * NX + ix - 1])
                - Cy * (u0[iy * NX + ix] - u0[(iy - 1) * NX + ix]);
        }
}

__global__ void initArray(float *u0, int NX, int NY, float dx, float dy)
{
    int ix0 = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int iy0 = blockIdx.y * blockDim.y + threadIdx.y + 1;

    int stride_x = blockDim.x * gridDim.x;
    int stride_y = blockDim.y * gridDim.y;

    for(int ix = ix0; ix < NX; ix += stride_x)
        for(int iy = iy0; iy < NY; iy += stride_y)
            u0[iy * NX + ix] = gaussian2D(p + (ix - 1) * dx,
                                          r + (iy - 1) * dy);
}

__global__ void exactSolution(float *u, int NX, int NY, float dx, float dy, float t)
{
    int ix0 = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int iy0 = blockIdx.y * blockDim.y + threadIdx.y + 1;

    int stride_x = blockDim.x * gridDim.x;
    int stride_y = blockDim.y * gridDim.y;

    for(int ix = ix0; ix < NX; ix += stride_x)
        for(int iy = iy0; iy < NY; iy += stride_y)
            u[iy * NX + ix] = gaussian2D(p + (ix - 1) * dx - vx * t,
                                         r + (iy - 1) * dy - vy * t);
}

void linearadvectionFOU2D(int NX, int NY, float *u0, float *u)
{
    /* spatial step size in x */
    float dx = (q - p) / (NX - 1);
    /* spatial step size in x */
    float dy = (s - r) / (NY - 1);
    /* initial time */
    float t = 0;
    /* safety factor */
    float F = 0.4;
    /* heuristic time step calc */
    float dt  = F * MIN(dx / fabs(vx), dy / fabs(vy));
    /* number of time steps */
    int Ntimesteps = MIN(500, 20 / dt);
    /* Courant number in the x direction */
    float Cx = dt * vx / dx;
    /* Courant number in the y direction */
    float Cy = dt * vy / dy;

    dim3 number_of_blocks(15, 16, 1);
    dim3 threads_per_block(16, 16, 1);

    /* Increase NX and NY to create an extra row and col with zero
     * as boundary conditions. */
    NX++; NY++;
    /* initial u vector, extended array for 'ghost values' */
    float *temp;
    initArray<<<number_of_blocks, threads_per_block>>>(u0, NX, NY, dx, dy);
    hipDeviceSynchronize();

    /* extended array for 'ghost values' mentioned below */
    for(int Ntimestep = 0; Ntimestep < Ntimesteps; Ntimestep++)
    {
        iteration<<<number_of_blocks, threads_per_block>>>(u, u0, Cx, Cy, NX, NY);
        hipDeviceSynchronize();
        temp = u0, u0 = u, u = temp;
        t += dt;
    }
    /* store the analytic solution in u */
    exactSolution<<<number_of_blocks, threads_per_block>>>(u, NX, NY, dx, dy, t);

    printf("Final t = %f\n", t);
}

void writetofile(float *u0, float *exact, int NX, int NY)
{
    NX++; NY++;
    /* Output */
    FILE* u0file = fopen("u0.txt", "w");
    FILE* exactfile = fopen("exact.txt", "w");
    FILE* difffile = fopen("diff.txt", "w");
    float Eu0, Eexact;
    for(int i = 1; i < NX; i++)
    {
        for(int j = 1; j < NY; j++)
        {
            Eu0 = u0[j * NX + i];
            Eexact = exact[j * NX + i];
            fprintf(u0file, "%10.6f ", Eu0);
            fprintf(exactfile, "%10.6f ", Eexact);
            fprintf(difffile, "%10.6f ", Eu0 - Eexact);
        }
        fprintf(u0file, "\n");
        fprintf(exactfile, "\n");
        fprintf(difffile, "\n");
    }
    fclose(u0file);
    fclose(exactfile);
    fclose(difffile);
}

int main()
{
    time_t start = clock();

    int NX = 2048;        /* number of grid points in x direction */
    int NY = 2048;        /* number of grid points in y direction */

    float *ud0, *ud, *uh0, *uh;
    /*cudaMallocManaged(&ud0, NX * NY, sizeof(float));*/
    /*cudaMallocManaged(&ud, NX * NY, sizeof(float));*/
    hipMalloc(&ud0, (NX + 1) * (NY + 1) * sizeof(float));
    hipMalloc(&ud, (NX + 1) * (NY + 1) * sizeof(float));

    linearadvectionFOU2D(NX, NY, ud0, ud);

    /* copy back to host memory */
    uh = (float*)malloc((NX + 1) * (NY + 1) * sizeof(float));
    uh0 = (float*)malloc((NX + 1) * (NY + 1) * sizeof(float));
    hipMemcpy(uh, ud, (NX + 1) * (NY + 1) * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(uh0, ud0, (NX + 1) * (NY + 1) * sizeof(float), hipMemcpyDeviceToHost);

    int testx = (int)(NX * 0.6);
    int testy = (int)(NY * 0.6);
    printf("u[%d][%d] = %f\n", testx, testy,
            uh0[(testx + 1) * (NX + 1) + testy + 1]);

    time_t end = clock();
    printf("Time: %f ms.\n", 1000.0 * (end - start) / CLOCKS_PER_SEC);

    printf("Writing to files ...\n");
    writetofile(uh0, uh, NX, NY);
    printf("Done.\n");

    hipFree(ud0);
    hipFree(ud);
    free(uh0);
    free(uh);

    return 0;
}
